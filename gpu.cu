#include <iostream>
#include "gpu.hpp"

void printCudaVersion()
{
    std::cout << "CUDA Compiled version: " << std::endl;
    std::cout << "\t__CUDACC_VER_MAJOR__:" << __CUDACC_VER_MAJOR__ << "\t";
    std::cout << "\t__CUDACC_VER_MINOR__:" << __CUDACC_VER_MINOR__ << "\t";
    std::cout << "\t__CUDACC_VER_BUILD__:" << __CUDACC_VER_BUILD__ << "\t";


    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}
